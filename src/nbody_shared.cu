#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 32
#define BLOCK_STRIDE 32

typedef struct
{
    float x, y, z, vx, vy, vz;
} Body;

void randomizeBodies(float *data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void bodyForce(Body *p, float dt, int n)
{

    // int i = threadIdx.x + blockIdx.x * blockDim.x;
    // 计算要处理的数据index
    int i = threadIdx.x + (int)(blockIdx.x / BLOCK_STRIDE) * blockDim.x;
    // 此块对应要处理的数据块的起始位置
    int start_block = blockIdx.x % BLOCK_STRIDE;
    if (i < n)
    {
        int cycle_times = n / BLOCK_SIZE;
        Body ptemp = p[i];
        // 使用shared_memory 多个线程读取同一块数据进入，提升存取性能
        __shared__ float3 spos[BLOCK_SIZE];
        Body temp;
        float dx, dy, dz, distSqr, invDist, invDist3;
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;
        // 这里的cycle_times 在已知块大小时使用常数性能会高一些
        for (int block_num = start_block; block_num < cycle_times; block_num += BLOCK_STRIDE)
        {
            temp = p[block_num * BLOCK_SIZE + threadIdx.x];
            spos[threadIdx.x] = make_float3(temp.x, temp.y, temp.z);
            // 块内同步，防止spos提前被读取
            __syncthreads();
            // 编译优化，只有 BLOCK_SIZE 为常量时才有用
#pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++)
            {
                dx = spos[j].x - ptemp.x;
                dy = spos[j].y - ptemp.y;
                dz = spos[j].z - ptemp.z;
                distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                invDist = rsqrtf(distSqr);
                invDist3 = invDist * invDist * invDist;
                Fx += dx * invDist3;
                Fy += dy * invDist3;
                Fz += dz * invDist3;
            }
            // 块内同步，防止spos提前被写入
            __syncthreads();
        }
        // 块之间不同步，原子加保证正确性
        atomicAdd(&p[i].vx, dt * Fx);
        atomicAdd(&p[i].vy, dt * Fy);
        atomicAdd(&p[i].vz, dt * Fz);
        // p[i].vx += dt * Fx;
        // p[i].vy += dt * Fy;
        // p[i].vz += dt * Fz;
    }
}

__global__ void integrate_position(Body *p, float dt, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv)
{

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1)
        nBodies = 2 << atoi(argv[1]);

    /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

    if (argc > 2)
        salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    hipHostMalloc(&buf, bytes);

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    double totalTime = 0.0;

    int deviceId;
    hipGetDevice(&deviceId);

    size_t threadsPerBlock = BLOCK_SIZE;
    size_t numberOfBlocks = (nBodies + threadsPerBlock - 1) / threadsPerBlock;

    float *d_buf;
    hipMalloc(&d_buf, bytes);
    Body *d_p = (Body *)d_buf;
    /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    /*******************************************************************/
    // Do not modify these 2 lines of code.gg
    for (int iter = 0; iter < nIters; iter++)
    {
        StartTimer();
    /*******************************************************************/

        /*
        * You will likely wish to refactor the work being done in `bodyForce`,
        * as well as the work to integrate the positions.
        */
        bodyForce<<<numberOfBlocks * BLOCK_STRIDE, threadsPerBlock>>>(d_p, dt, nBodies); // compute interbody forces
        /*
        * This position integration cannot occur until this round of `bodyForce` has completed.
        * Also, the next round of `bodyForce` cannot begin until the integration is complete.
        */
        integrate_position<<<nBodies / threadsPerBlock, threadsPerBlock>>>(d_p, dt, nBodies);

        if (iter == nIters - 1)
        {
            hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
        }

    /*******************************************************************/
    // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
   * Feel free to modify code below.
   */
    hipFree(d_buf);
    hipHostFree(buf);
}
