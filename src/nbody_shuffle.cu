#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 32
#define BLOCK_STRIDE 32

typedef struct
{
    float x, y, z, vx, vy, vz;
} Body;

void randomizeBodies(float *data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void bodyForce(Body *p, float dt, int n)
{

    // int i = threadIdx.x + blockIdx.x * blockDim.x;
    int cycle_times = n / BLOCK_SIZE;
    // 计算要处理的数据index
    int i = threadIdx.x + (int)(blockIdx.x / BLOCK_STRIDE) * blockDim.x;
    // 此块对应要处理的数据块的起始位置
    int start_block = blockIdx.x % BLOCK_STRIDE;
    if (i < n)
    {
        Body ptemp = p[i];
        Body temp;
        float share_x,share_y,share_z;
        float dx, dy, dz, distSqr, invDist, invDist3;
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;
        // 这里的cycle_times 在已知块大小时使用常数性能会高一些
        for (int block_num = start_block; block_num < cycle_times; block_num += BLOCK_STRIDE)
        {
            temp = p[block_num * BLOCK_SIZE + threadIdx.x];
            share_x = temp.x;
            share_y = temp.y;
            share_z = temp.z;
            // 编译优化，只有 BLOCK_SIZE 为常量时才有用
#pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++)
            {
                dx = __shfl_sync(0xFFFFFFFF,share_x,j) - ptemp.x;
                dy = __shfl_sync(0xFFFFFFFF,share_y,j) - ptemp.y;
                dz = __shfl_sync(0xFFFFFFFF,share_z,j) - ptemp.z;
                distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                invDist = rsqrtf(distSqr);
                invDist3 = invDist * invDist * invDist;
                Fx += dx * invDist3;
                Fy += dy * invDist3;
                Fz += dz * invDist3;
            }
            // 块内同步，防止spos提前被写入
            __syncthreads();
        }
        // 块之间不同步，原子加保证正确性
        atomicAdd(&p[i].vx, dt * Fx);
        atomicAdd(&p[i].vy, dt * Fy);
        atomicAdd(&p[i].vz, dt * Fz);
        // p[i].vx += dt * Fx;
        // p[i].vy += dt * Fy;
        // p[i].vz += dt * Fz;
    }
}

__global__ void integrate_position(Body *p, float dt, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv)
{

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1)
        nBodies = 2 << atoi(argv[1]);

    /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

    if (argc > 2)
        salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    hipHostMalloc(&buf, bytes);

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    double totalTime = 0.0;

    int deviceId;
    hipGetDevice(&deviceId);

    size_t threadsPerBlock = BLOCK_SIZE;
    size_t numberOfBlocks = (nBodies + threadsPerBlock - 1) / threadsPerBlock;

    float *d_buf;
    hipMalloc(&d_buf, bytes);
    Body *d_p = (Body *)d_buf;
    /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    /*******************************************************************/
    // Do not modify these 2 lines of code.gg
    for (int iter = 0; iter < nIters; iter++)
    {
        StartTimer();
    /*******************************************************************/

        /*
        * You will likely wish to refactor the work being done in `bodyForce`,
        * as well as the work to integrate the positions.
        */
        bodyForce<<<numberOfBlocks * BLOCK_STRIDE, threadsPerBlock>>>(d_p, dt, nBodies); // compute interbody forces
        /*
        * This position integration cannot occur until this round of `bodyForce` has completed.
        * Also, the next round of `bodyForce` cannot begin until the integration is complete.
        */
        integrate_position<<<nBodies / threadsPerBlock, threadsPerBlock>>>(d_p, dt, nBodies);

        if (iter == nIters - 1)
        {
            hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
        }

    /*******************************************************************/
    // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
   * Feel free to modify code below.
   */
    hipFree(d_buf);
    hipHostFree(buf);
}
